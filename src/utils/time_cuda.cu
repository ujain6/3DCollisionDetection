
#include "time_cuda.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


void start_cuda_timing(int time_id){
    hipError_t error;

    hipEvent_t * start = (hipEvent_t *) malloc(sizeof(hipEvent_t));
    error = hipEventCreate(start);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    GTIMES_BUF[time_id] = (timestruct_t) start;
    // Record the start event
    error = hipEventRecord(*start, NULL);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }   
}

void stop_cuda_timing(int time_id){
    hipEvent_t * start = (hipEvent_t *) GTIMES_BUF[time_id]; // Retrieve start event
    hipEvent_t stop;
    
    hipError_t error;
    error = hipEventCreate(&stop);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }   
    // Record the stop event
    error = hipEventRecord(stop, NULL);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess){
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, *start, stop);

    if (error != hipSuccess){
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    GTIMES[time_id] = msecTotal;
    free(start);
}